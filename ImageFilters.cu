#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <windows.h>


using namespace std;

BITMAPFILEHEADER bitfilehead; // 14 bajtowy nag³ówek pliku bitmapy,/ Zawiera informacje o parametrach pliku BMP
BITMAPINFOHEADER bitinfohead; // Zawiera informacje na temat wymiarów i kolorów w formacie DIB,/ Zawiera informacje o parametrach bitmapy.

// Wczytywanie nag³ówka pliku
void OpenImage()
{
	char *filename = "lena.bmp";
	FILE *input = fopen(filename, "rb+");

	if (input == NULL)
	{
		printf("- Plik nie zostal otwarty (Wczytywanie naglowka pliku)\n");
		exit(0);
	}
	printf("- Plik zostal otwarty pomyslnie (Wczytywanie naglowka pliku)\n");

	if (fread(&bitfilehead, sizeof(BITMAPFILEHEADER), 1, input) != 1) {
		printf(" Blad w odczycie naglowka bmp\n");
	}

	if (fread(&bitinfohead, sizeof(BITMAPINFOHEADER), 1, input) != 1) {
		printf(" Blad w odczycie informacji o zdjeciu\n");
	}
	fclose(input);

}


__global__ void GPUlowFilter(unsigned char* buffer, unsigned char* result, int width, int height)
{
	int col = threadIdx.x;
	int row = blockIdx.x * width * 3;
	const int maskSize = 7;
	int maskSizeHalf = (maskSize - 1) / 2;
	if (blockIdx.x >= height - maskSize-2)

		return;
	//Maska 5x5 Filtr dolnoprzepustowy
	/*int mask[maskSize][maskSize] = { 1, 1, 1, 1, 1,
									   1, 1, 1, 1, 1,
									   1, 1, 1, 1, 1,
									   1, 1, 1, 1, 1,
									   1, 1, 1, 1, 1 };*/
	int mask[maskSize][maskSize] = { 1, 1, 2, 2 , 2, 1, 1,
									 1, 2, 2, 4 , 2, 2, 1,
									 2, 2, 4, 8 , 4, 2, 2,
									 2, 4, 8, 16, 8, 4, 2,
									 2, 2, 4, 8 , 4, 2, 2,
									 1, 2, 2, 4 , 2, 2, 1,
									 1, 1, 2, 2 , 2, 1, 1,};
	//Maska 17x17 Filtr dolnoprzepustowy
	 /*int mask[maskSize][maskSize] =	   {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,};*/

	//Suma wszystkich argumentow maski
	// int devider = 289; //dla 17x17
	int devider = 120;
	for (int c = 0; c < 3; c++){
		
		if (c == 3 && col >= width - maskSize-2){
			return;
		}
	
		for (int channel = 0; channel < 3; channel++){
			float sum = 0.0;
			for (int j = -maskSizeHalf; j <= maskSizeHalf; j++){

				for (int i = -maskSizeHalf; i <= maskSizeHalf; i++){

					int rowN = (blockIdx.x + j) * width * 3;
					int  color = buffer[(rowN)+(col + i) + width*c + channel];//channel0
					sum += color * mask[i + maskSizeHalf][j + maskSizeHalf];
				}
			}
			result[(row + col) + width*c+channel] = (char)(sum / devider);
		}
	}
}
__global__ void GPUhightFilter(unsigned char* buffer, unsigned char* result, int width, int height)
{
	int col = threadIdx.x;
	int row = blockIdx.x * width * 3;
	const int maskSize = 3;
	int maskSizeHalf = (maskSize - 1) / 2;
	if (blockIdx.x >= height - maskSize - 2)

	return;
	//Maska 3x3 Filtr gornoprzepustowy
	int mask[maskSize][maskSize] = {  1, -2,  1,
									 -2,  5, -2,
								      1, -2,  1, };
	//Maska 5x5 Filtr gornoprzepustowy
	/*int mask[maskSize][maskSize] = { -1, -1, -1, -1, -1,
									 -1, -1, -1, -1, -1,
									 -1, -1, 24, -1, -1,
									 -1, -1, -1, -1, -1,
									 -1, -1, -1, -1, -1 };*/

	//Maska 7x7 Filtr gornoprzepustowy
	/*int mask[maskSize][maskSize] = { -1, -1, -1, -1, -1, -1, -1, 
									 -1, -1, -1, -1, -1, -1, -1, 
									 -1, -1, -1, -1, -1, -1, -1, 
									 -1, -1, -1, 48, -1, -1, -1, 
									 -1, -1, -1, -1, -1, -1, -1, 
									 -1, -1, -1, -1, -1, -1, -1, 
									 -1, -1, -1, -1, -1, -1, -1,};*/

	//Suma wszystkich argumentow maski
	int devider = 1;

	for (int c = 0; c < 3; c++){

		if (c == 3 && col >= width - maskSize - 2){
			return;
		}

		for (int channel = 0; channel < 3; channel++){
			float sum = 0.0;
			for (int j = -maskSizeHalf; j <= maskSizeHalf; j++){

				for (int i = -maskSizeHalf; i <= maskSizeHalf; i++){

					int rowN = (blockIdx.x + j) * width * 3;
					int  color = buffer[(rowN)+(col + i) + width*c + channel];//channel0
					sum += color * mask[i + maskSizeHalf][j + maskSizeHalf];
				}
			}
			result[(row + col) + width*c + channel] = (char)(sum / devider);
		}
	}
}



int main()
{
	
	int start = GetTickCount(); // Pobiera aktualny czas

	OpenImage();



	int channels = bitinfohead.biBitCount/ 8;

	unsigned long int n = bitinfohead.biWidth*bitinfohead.biHeight * channels;
	
	
	unsigned char *buffer_cuda;
	unsigned char *result_cuda;
	
	
	hipMalloc((void**)&buffer_cuda, bitinfohead.biWidth*bitinfohead.biHeight * channels * sizeof(unsigned char));
	hipMalloc((void**)&result_cuda, bitinfohead.biWidth*bitinfohead.biHeight * channels * sizeof(unsigned char));
	
	unsigned char *buffer = (unsigned char*)malloc(bitinfohead.biWidth*bitinfohead.biHeight * channels);
	unsigned char *result = (unsigned char*)malloc(bitinfohead.biWidth*bitinfohead.biHeight * channels);
	
	//Czytanie danych z pliku i tworzenie chara z danymi zdjêcia
	char *filename = "lena.bmp";
	FILE *input = fopen(filename, "rb+");

	fseek(input, bitfilehead.bfOffBits, SEEK_SET);

	for (int i = 0; i < n; i++)
	{
		buffer[i] = fgetc(input);
	}
	printf("- Odczytano pomyslnie zawartosc danych o obrazie i zapisano do tablicy na CPU\n");
	fclose(input);

	hipMemcpy(buffer_cuda, buffer, bitinfohead.biWidth*bitinfohead.biHeight * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
	
	int choice;
	cout << "Filtr" << endl;
	cout << " LowPass Filter [1] " << endl;
	cout << " HightPass Filter [2] " << endl;
	cin >> choice;
	switch (choice)
	{
	case 1: {
				

				GPUlowFilter << <512, 512 >> > (buffer_cuda, result_cuda, bitinfohead.biWidth, bitinfohead.biHeight * 3);
				
	}
		break;
	case 2: 

				GPUhightFilter << <512, 512 >> > (buffer_cuda, result_cuda, bitinfohead.biWidth, bitinfohead.biHeight * 3);
		break;
	}
	


	hipMemcpy(result, result_cuda, bitinfohead.biWidth*bitinfohead.biHeight * channels, hipMemcpyDeviceToHost);
	
	//Zapisywanie wyjœciowych danych z kernela jako plik bmp
	char *plik = "projekt_output.bmp";
	FILE *output = fopen(plik, "wb+");

	if (output == NULL)
	{
		printf("- Plik nie zostal otwarty (Zapisywanie bitmapy do pliku)\n");
		exit(0);
	}
	printf("- Plik zostal otwarty pomyslnie (Zapisywanie bitmapy do pliku)\n");

	// Zapis nag³ówka
	fwrite(&bitfilehead, 1, sizeof(bitfilehead), output);
	// Zapis informacji o pliku
	fwrite(&bitinfohead, sizeof(bitinfohead), 1, output);
	// Zapisuje dane obrazu
	fwrite(result, sizeof(unsigned char), bitinfohead.biWidth*bitinfohead.biHeight * channels, output);

	fclose(output);


	hipFree(buffer_cuda);
	hipFree(result_cuda);
	free(result);
	free(buffer);

	cout << "Czas wykonania kodu aplikacji: " << GetTickCount() - start << "ms." << endl;
	system("pause");
	return 0;
}
